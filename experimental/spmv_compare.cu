/******************************************************************************
 * Copyright (c) 2011-2015, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

//---------------------------------------------------------------------
// SpMV comparison tool
//---------------------------------------------------------------------

#include <stdio.h>
#include <map>
#include <vector>
#include <algorithm>
#include <cstdio>
#include <fstream>

#include <hipsparse.h>

#include "matrix.h"

#include <cub/util_allocator.cuh>
#include <test/test_util.h>

using namespace hipcub;


//---------------------------------------------------------------------
// Globals, constants, and type declarations
//---------------------------------------------------------------------

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

bool                    g_verbose = false;  // Whether to display input/output to console
CachingDeviceAllocator  g_allocator(true);  // Caching allocator for device memory


//---------------------------------------------------------------------
// SpMV verification
//---------------------------------------------------------------------

// Compute reference SpMV y = Ax
template <
    typename VertexT,
    typename ValueT,
    typename SizeT>
void SpmvGold(
    CsrMatrix<VertexT, ValueT, SizeT>&      matrix_a,
    ValueT*                                 vector_x,
    ValueT*                                 vector_y)
{
    for (SizeT row = 0; row < matrix_a.num_rows; ++row)
    {
        vector_y[row] = 0;
        for (
            SizeT column = matrix_a.row_offsets[row];
            column < matrix_a.row_offsets[row + 1];
            ++column)
        {
            vector_y[row] += matrix_a.values[column] * vector_x[matrix_a.column_indices[column]];
        }
    }
}


//---------------------------------------------------------------------
// Test GPU SpMV execution
//---------------------------------------------------------------------

/**
 * Run cuSparse SpMV (specialized for fp32)
 */
template <
    typename VertexT,
    typename SizeT>
float CusparseSpmv(
    int                 num_rows,
    int                 num_cols,
    int                 num_nonzeros,
    float*              d_matrix_values,
    SizeT*              d_matrix_row_offsets,
    VertexT*            d_matrix_column_indices,
    float*              d_vector_x,
    float*              d_vector_y,
    int                 timing_iterations,
    hipsparseHandle_t    cusparse)
{
    hipsparseMatDescr_t desc;
    hipsparseCreateMatDescr(&desc);
    float alpha             = 1.0;
    float beta              = 0.0;

    float elapsed_millis    = 0.0;
    GpuTimer gpu_timer;

    for(int it = 0; it < timing_iterations; ++it)
    {
        gpu_timer.Start();

        hipsparseStatus_t status = hipsparseScsrmv(
            cusparse,
            HIPSPARSE_OPERATION_NON_TRANSPOSE,
            num_rows,
            num_cols,
            num_nonzeros,
            &alpha,
            desc,
            d_matrix_values,
            d_matrix_row_offsets,
            d_matrix_column_indices,
            d_vector_x,
            &beta,
            d_vector_y);

        gpu_timer.Stop();
        elapsed_millis += gpu_timer.ElapsedMillis();
    }

    hipsparseDestroyMatDescr(desc);
    return elapsed_millis / timing_iterations;
}


/**
 * Run cuSparse SpMV (specialized for fp64)
 */
template <
    typename VertexT,
    typename SizeT>
float CusparseSpmv(
    int                 num_rows,
    int                 num_cols,
    int                 num_nonzeros,
    double*             d_matrix_values,
    SizeT*              d_matrix_row_offsets,
    VertexT*            d_matrix_column_indices,
    double*             d_vector_x,
    double*             d_vector_y,
    int                 timing_iterations,
    hipsparseHandle_t    cusparse)
{
    hipsparseMatDescr_t desc;
    hipsparseCreateMatDescr(&desc);
    double alpha            = 1.0;
    double beta             = 0.0;

    float elapsed_millis    = 0.0;
    GpuTimer gpu_timer;

    for(int it = 0; it < timing_iterations; ++it)
    {
        gpu_timer.Start();

        hipsparseStatus_t status = hipsparseDcsrmv(
            cusparse,
            HIPSPARSE_OPERATION_NON_TRANSPOSE,
            num_rows,
            num_cols,
            num_nonzeros,
            &alpha,
            desc,
            d_matrix_values,
            d_matrix_row_offsets,
            d_matrix_column_indices,
            d_vector_x,
            &beta,
            d_vector_y);

        gpu_timer.Stop();
        elapsed_millis += gpu_timer.ElapsedMillis();
    }

    hipsparseDestroyMatDescr(desc);
    return elapsed_millis / timing_iterations;
}


/**
 * Run tests
 */
template <
    typename VertexT,
    typename ValueT,
    typename SizeT>
void RunTests(
    std::string         &mtx_filename,
    int                 grid2d,
    int                 grid3d,
    int                 wheel,
    int                 timing_iterations,
    float               bandwidth_GBs,
    hipsparseHandle_t    cusparse)
{
    // Initialize matrix in COO form
    CooMatrix<VertexT, ValueT> coo_matrix;

    if (!mtx_filename.empty())
    {
        // Parse matrix market file
        cout << "Reading matrix market file " << mtx_filename << "... "; fflush(stdout);
        coo_matrix.InitMarket(mtx_filename);
        cout << "done.\n"; fflush(stdout);
    }
    else if (grid2d > 0)
    {
        // Generate 2D lattice
        coo_matrix.InitGrid2d(grid2d, false);
    }
    else if (grid3d > 0)
    {
        // Generate 3D lattice
        coo_matrix.InitGrid3d(grid3d, false);
    }
    else if (wheel > 0)
    {
        // Generate wheel graph
        coo_matrix.InitWheel(wheel);
    }
    else
    {
        fprintf(stderr, "No graph type specified.\n");
        exit(1);
    }

    CsrMatrix<VertexT, ValueT, SizeT> csr_matrix;
    csr_matrix.FromCoo(coo_matrix);

    // Display matrix info
    csr_matrix.DisplayHistogram();

    // Allocate input and output vectors
    ValueT* vector_x = new ValueT[csr_matrix.num_cols];
    ValueT* vector_y = new ValueT[csr_matrix.num_rows];

    for (int col = 0; col < csr_matrix.num_cols; ++col)
        vector_x[col] = 1.0;

    // Compute reference answer
    SpmvGold(csr_matrix, vector_x, vector_y);

    // Allocate and initialize GPU problem
    ValueT*             d_matrix_values;
    SizeT*              d_matrix_row_offsets;
    VertexT*            d_matrix_column_indices;
    ValueT*             d_vector_x;
    ValueT*             d_vector_y;

    g_allocator.DeviceAllocate((void **) &d_matrix_values,          sizeof(ValueT) * csr_matrix.num_nonzeros);
    g_allocator.DeviceAllocate((void **) &d_matrix_row_offsets,     sizeof(SizeT) * (csr_matrix.num_rows + 1));
    g_allocator.DeviceAllocate((void **) &d_matrix_column_indices,  sizeof(VertexT) * csr_matrix.num_nonzeros);
    g_allocator.DeviceAllocate((void **) &d_vector_x,               sizeof(ValueT) * csr_matrix.num_cols);
    g_allocator.DeviceAllocate((void **) &d_vector_y,               sizeof(ValueT) * csr_matrix.num_rows);

    CubDebugExit(hipMemcpy(d_matrix_values,            csr_matrix.values,          sizeof(ValueT) * csr_matrix.num_nonzeros, hipMemcpyHostToDevice));
    CubDebugExit(hipMemcpy(d_matrix_row_offsets,       csr_matrix.row_offsets,     sizeof(SizeT) * (csr_matrix.num_rows + 1), hipMemcpyHostToDevice));
    CubDebugExit(hipMemcpy(d_matrix_column_indices,    csr_matrix.column_indices,  sizeof(VertexT) * csr_matrix.num_nonzeros, hipMemcpyHostToDevice));
    CubDebugExit(hipMemcpy(d_vector_x,                 vector_x,                   sizeof(ValueT) * csr_matrix.num_cols, hipMemcpyHostToDevice));

    double avg_millis, nz_throughput, effective_bandwidth;
    int compare = 0;
    size_t total_bytes = (csr_matrix.num_nonzeros * (sizeof(ValueT) * 2 + sizeof(VertexT))) +
        (csr_matrix.num_rows) * (sizeof(SizeT) + sizeof(ValueT));

    // Run problem on cuSparse

    CubDebugExit(hipMemset(d_vector_y, 0, sizeof(ValueT) * csr_matrix.num_rows));

    avg_millis = CusparseSpmv(csr_matrix.num_rows, csr_matrix.num_cols,
        csr_matrix.num_nonzeros, d_matrix_values, d_matrix_row_offsets,
        d_matrix_column_indices, d_vector_x, d_vector_y, timing_iterations,
        cusparse);

    nz_throughput       = double(csr_matrix.num_nonzeros) / avg_millis / 1.0e6;
    effective_bandwidth = double(total_bytes) / avg_millis / 1.0e6;

    printf("%s fp%d: %.3f avg ms, %.3f gflops, %.3lf effective GB/s (%.1f%% peak)\n",
        "cuSparse",
        sizeof(ValueT) * 8,
        avg_millis,
        2 * nz_throughput,
        effective_bandwidth,
        effective_bandwidth / bandwidth_GBs * 100);

    compare = CompareDeviceResults(vector_y, d_vector_y, csr_matrix.num_rows, true, g_verbose);
    printf("\t%s\n", compare ? "FAIL" : "PASS"); fflush(stdout);
    AssertEquals(0, compare);

    // Cleanup
    delete[] vector_x;
    delete[] vector_y;
}



/**
 * Main
 */
int main(int argc, char **argv)
{
    // Initialize command line
    CommandLineArgs args(argc, argv);
    if (args.CheckCmdLineFlag("help"))
    {
        printf(
            "%s "
            "[--device=<device-id>] "
            "[--v] "
            "[--i=<timing iterations>] "
            "[--fp64] "
            "\n\t"
                "--mtx=<matrix market file> "
            "\n\t"
                "--grid2d=<width>"
            "\n\t"
                "--grid3d=<width>"
            "\n\t"
                "--wheel=<spokes>"
            "\n", argv[0]);
        exit(0);
    }

    bool                fp64;
    std::string         mtx_filename;
    int                 grid2d              = -1;
    int                 grid3d              = -1;
    int                 wheel               = -1;
    int                 timing_iterations   = 100;

    g_verbose = args.CheckCmdLineFlag("v");
    fp64 = args.CheckCmdLineFlag("fp64");
    args.GetCmdLineArgument("i", timing_iterations);
    args.GetCmdLineArgument("mtx", mtx_filename);
    args.GetCmdLineArgument("grid2d", grid2d);
    args.GetCmdLineArgument("grid3d", grid3d);
    args.GetCmdLineArgument("wheel", wheel);

    // Initialize device
    CubDebugExit(args.DeviceInit());

    // Initalize cuSparse
    hipsparseHandle_t cusparse;
    AssertEquals(HIPSPARSE_STATUS_SUCCESS, hipsparseCreate(&cusparse));

    // Get GPU device bandwidth (GB/s)
    int device_ordinal, bus_width, mem_clock_khz;
    CubDebugExit(hipGetDevice(&device_ordinal));
    CubDebugExit(hipDeviceGetAttribute(&bus_width, hipDeviceAttributeMemoryBusWidth, device_ordinal));
    CubDebugExit(hipDeviceGetAttribute(&mem_clock_khz, hipDeviceAttributeMemoryClockRate, device_ordinal));
    float bandwidth_GBs = float(bus_width) * mem_clock_khz * 2 / 8 / 1000 / 1000;

    // Run test(s)
    if (fp64)
    {
        RunTests<int, double, int>(mtx_filename, grid2d, grid3d, wheel, timing_iterations, bandwidth_GBs, cusparse);
    }
    else
    {
        RunTests<int, float, int>(mtx_filename, grid2d, grid3d, wheel, timing_iterations, bandwidth_GBs, cusparse);
    }

    CubDebugExit(hipDeviceSynchronize());
    printf("\n\n");

    return 0;
}
