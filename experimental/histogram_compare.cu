#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011-2015, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include <stdio.h>
#include <map>
#include <vector>
#include <algorithm>

#include "histogram/readers.h"
#include "histogram/histogram_gmem_atomics.h"
#include "histogram/histogram_smem_atomics.h"
#include "histogram/histogram_cub.h"

/*
#include "histogram/histogram_smem_write.h"
#include "histogram/histogram_no_atomics.h"
#include "histogram/histogram_smem_warp_coalescing.h"
*/

#include <cub/util_allocator.cuh>
#include <cub/device/device_scan.cuh>
#include <test/test_util.h>

using namespace hipcub;

//---------------------------------------------------------------------
// Globals, constants, and type declarations
//---------------------------------------------------------------------

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

bool                    g_verbose = false;  // Whether to display input/output to console
CachingDeviceAllocator  g_allocator(true);  // Caching allocator for device memory

struct less_than_value
{
    inline bool operator()(
        const std::pair<std::string, double> &a,
        const std::pair<std::string, double> &b)
    {
        return a.second < b.second;
    }
};


//---------------------------------------------------------------------
// Test generation
//---------------------------------------------------------------------


// Compute reference histogram.
template <
    int         ACTIVE_CHANNELS,
    int         NUM_BINS,
    typename    PixelType>
void histogram_gold(PixelType *image, int width, int height, unsigned int* hist);


// Compute reference histogram.  Specialized for uchar4
template <
    int         ACTIVE_CHANNELS,
    int         NUM_BINS>
void histogram_gold(uchar4 *image, int width, int height, unsigned int* hist)
{
    for (int i = 0; i < width; i++)
    {
        for (int j = 0; j < height; j++)
        {
            uchar4 pixel = image[i + j * width];

            unsigned int r_bin = (unsigned int) pixel.x;
            unsigned int g_bin = (unsigned int) pixel.y;
            unsigned int b_bin = (unsigned int) pixel.z;
            unsigned int a_bin = (unsigned int) pixel.w;

            if (ACTIVE_CHANNELS > 0)
                hist[(NUM_BINS * 0) + r_bin]++;
            if (ACTIVE_CHANNELS > 1)
                hist[(NUM_BINS * 1) + g_bin]++;
            if (ACTIVE_CHANNELS > 2)
                hist[(NUM_BINS * 2) + b_bin]++;
            if (ACTIVE_CHANNELS > 3)
                hist[(NUM_BINS * 3) + a_bin]++;
        }
    }
}


// Compute reference histogram.  Specialized for float4
template <
    int         ACTIVE_CHANNELS,
    int         NUM_BINS>
void histogram_gold(float4 *image, int width, int height, unsigned int* hist)
{
    memset(hist, 0, ACTIVE_CHANNELS * NUM_BINS * sizeof(unsigned int));

    for (int i = 0; i < width; i++)
    {
        for (int j = 0; j < height; j++)
        {
            float4 pixel = image[i + j * width];

            unsigned int r_bin = (unsigned int) (pixel.x * NUM_BINS);
            unsigned int g_bin = (unsigned int) (pixel.y * NUM_BINS);
            unsigned int b_bin = (unsigned int) (pixel.z * NUM_BINS);
            unsigned int a_bin = (unsigned int) (pixel.w * NUM_BINS);

            if (ACTIVE_CHANNELS > 0)
                hist[(NUM_BINS * 0) + r_bin]++;
            if (ACTIVE_CHANNELS > 1)
                hist[(NUM_BINS * 1) + g_bin]++;
            if (ACTIVE_CHANNELS > 2)
                hist[(NUM_BINS * 2) + b_bin]++;
            if (ACTIVE_CHANNELS > 3)
                hist[(NUM_BINS * 3) + a_bin]++;
        }
    }
}


/**
 * Run a specific histogram implementation
 */
template <
    int         ACTIVE_CHANNELS,
    int         NUM_BINS,
    typename    PixelType>
void RunTest(
    std::vector<std::pair<std::string, double> >&   timings,
    PixelType*                                      d_pixels,
    const int                                       width,
    const int                                       height,
    unsigned int *                                  d_hist,
    unsigned int *                                  h_hist,
    int                                             timing_iterations,
    const char *                                    long_name,
    const char *                                    short_name,
    double (*f)(PixelType*, int, int, unsigned int*))
{
    printf("%s ", long_name);
    double elapsed_time = 0;
    for (int i = 0; i < timing_iterations; i++)
    {
        elapsed_time += (*f)(d_pixels, width, height, d_hist);
    }
    double avg_time = elapsed_time /= timing_iterations;    // average
    timings.push_back(std::pair<std::string, double>(short_name, avg_time));

    printf("Avg time %.3f ms (%d iterations)\n", avg_time, timing_iterations);

    int compare = CompareDeviceResults(h_hist, d_hist, ACTIVE_CHANNELS * NUM_BINS, true, g_verbose);
    printf("\t%s\n", compare ? "FAIL" : "PASS");
    AssertEquals(0, compare);
}


/**
 * Evaluate a variety of different histogram implementations
 */
template <
    int         ACTIVE_CHANNELS,
    int         NUM_BINS,
    typename    PixelType>
void RunTests(
    PixelType*  h_pixels,
    int         height,
    int         width,
    int         timing_iterations)
{
    // Copy data to gpu
    PixelType* d_pixels;
    size_t pixel_bytes = width * height * sizeof(PixelType);
    CubDebugExit(g_allocator.DeviceAllocate((void**) &d_pixels, pixel_bytes));
    CubDebugExit(hipMemcpy(d_pixels, h_pixels, pixel_bytes, hipMemcpyHostToDevice));

    // Allocate results arrays on cpu/gpu
    unsigned int *h_hist;
    unsigned int *d_hist;
    size_t channel_bytes = NUM_BINS * sizeof(unsigned int);
    h_hist = (unsigned int *) malloc(channel_bytes * ACTIVE_CHANNELS);
    g_allocator.DeviceAllocate((void **) &d_hist, channel_bytes * ACTIVE_CHANNELS);

    // Compute reference cpu histogram
    histogram_gold<ACTIVE_CHANNELS, NUM_BINS>(h_pixels, width, height, h_hist);

    // Store timings
    std::vector<std::pair<std::string, double> > timings;

    // Run experiments
    RunTest<ACTIVE_CHANNELS, NUM_BINS>(timings, d_pixels, width, height, d_hist, h_hist, timing_iterations,
        "Global memory atomics", "gmem atomics", run_gmem_atomics<ACTIVE_CHANNELS, NUM_BINS, PixelType>);
    RunTest<ACTIVE_CHANNELS, NUM_BINS>(timings, d_pixels, width, height, d_hist, h_hist, timing_iterations,
        "Shared memory atomics", "smem atomics", run_smem_atomics<ACTIVE_CHANNELS, NUM_BINS, PixelType>);
    RunTest<ACTIVE_CHANNELS, NUM_BINS>(timings, d_pixels, width, height, d_hist, h_hist, timing_iterations,
        "CUB", "CUB", run_cub_histogram<ACTIVE_CHANNELS, NUM_BINS, PixelType>);

/*
    run_experiment(timings, d_pixels, width, height, d_hist,
        h_hist, "Shared memory atomics", "smem atomics", run_smem_atomics);
    run_experiment(timings, d_pixels, width, height, d_hist,
        h_hist, "No atomics (NPP)", "no atomics",
        run_no_atomics);
    run_experiment(timings, d_pixels, width, height, d_hist,
        h_hist, "Warp coalescing", "warp coalescing", run_smem_warp_coalescing);
    run_experiment(timings, d_pixels, width, height, d_hist,
        h_hist, "CUB histogram atomics", "cub", run_cub);
*/

    // Report timings
    std::sort(timings.begin(), timings.end(), less_than_value());
    printf("Timings (ms):\n");
    for (int i = 0; i < timings.size(); i++)
        printf("  %.3f %s\n", timings[i].second, timings[i].first.c_str());

    // Free data
    CubDebugExit(g_allocator.DeviceFree(d_pixels));
    CubDebugExit(g_allocator.DeviceFree(d_hist));
    free(h_hist);
}


/**
 * Main
 */
int main(int argc, char **argv)
{
    enum {
        ACTIVE_CHANNELS = 3,
        NUM_BINS        = 256

    };

    // Initialize command line
    CommandLineArgs     args(argc, argv);
    int                 timing_iterations   = 100;
    std::string         filename;
    int                 height              = -1;
    int                 width               = -1;
    g_verbose = args.CheckCmdLineFlag("v");
    args.GetCmdLineArgument("i", timing_iterations);
    args.GetCmdLineArgument("file", filename);
    args.GetCmdLineArgument("height", height);
    args.GetCmdLineArgument("width", width);

    // Print usage
    if (args.CheckCmdLineFlag("help"))
    {
        printf("%s "
            "[--device=<device-id>] "
            "[--v] "
            "[--i=<timing iterations>] "
            "--file=<filename.[tga|bin]> "
            "[--height=<binfile height>] "
            "[--width=<binfile width>] "
            "\n", argv[0]);
        exit(0);
    }

    // Initialize device
    CubDebugExit(args.DeviceInit());

    if (filename.find(".tga") != std::string::npos)
    {
        // Parse targa file
        printf("Targa (uchar4):\n");
        uchar4* byte_pixels = NULL;
        ReadTga(byte_pixels, width, height, filename.c_str());

        // uchar4 tests
//        RunTests<ACTIVE_CHANNELS, NUM_BINS>(byte_pixels, width, height, timing_iterations);

        // Convert uchar4 to float4 pixels
        float4* float_pixels = NULL;
        if ((float_pixels = (float4*) malloc(width * height * sizeof(float4))) == NULL)
        {
            fprintf(stderr, "malloc of image failed\n");
            exit(-1);
        }
        for (int i = 0; i < width * height; ++i)
        {
            float_pixels[i].x = ((float) byte_pixels[i].x) / NUM_BINS;
            float_pixels[i].y = ((float) byte_pixels[i].y) / NUM_BINS;
            float_pixels[i].z = ((float) byte_pixels[i].z) / NUM_BINS;
            float_pixels[i].w = ((float) byte_pixels[i].w) / NUM_BINS;
        }

        // float4 tests
        RunTests<ACTIVE_CHANNELS, NUM_BINS>(float_pixels, width, height, timing_iterations);

        // Free pixel data
        free(byte_pixels);
        free(float_pixels);
    }
    else if (filename.find(".bin") != std::string::npos)
    {
        // Parse float4 binary file
        printf("Bin (float4):\n");
        float4* float_pixels = NULL;
        ReadBin(float_pixels, width, height, filename.c_str());
        RunTests<ACTIVE_CHANNELS, NUM_BINS>(float_pixels, width, height, timing_iterations);
        free(float_pixels);
    }

    CubDebugExit(hipDeviceSynchronize());
    printf("\n\n");

    return 0;
}
